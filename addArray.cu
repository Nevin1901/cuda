
#include <hip/hip_runtime.h>
#include <iostream>
#include <X11/Xlib.h>
#include <unistd.h>

__global__ void populateVec(int n, float *a, float *b) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int stride_y = blockDim.y * gridDim.y;

    for (int i = index; i < n; i+= stride) {
        a[i] = 2;
        b[i] = 5;
    }
}

__global__ void vecAdd(int n, float *a, float *b, float *c) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i+= stride) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    Display *dpy = XOpenDisplay(0);

    int blackColor = BlackPixel(dpy, DefaultScreen(dpy));
    int whiteColor = WhitePixel(dpy, DefaultScreen(dpy));

    Window w = XCreateSimpleWindow(dpy, DefaultRootWindow(dpy), 0, 0, 200, 100, 0, blackColor, blackColor);

    XSelectInput(dpy, w, StructureNotifyMask);

    XMapWindow(dpy, w);

    GC gc = XCreateGC(dpy, w, 0, 0);

    XSetForeground(dpy, gc, whiteColor);

    for (;;) {
        XEvent e;
        XNextEvent(dpy, &e);
        if (e.type == MapNotify) {
            break;
        }
    }

    XDrawLine(dpy, w, gc, 10, 60, 180, 20);

    XFlush(dpy);

    sleep(10);
    // int nDevices;

    // cudaGetDeviceCount(&nDevices);

    // for (int k = 0; k < nDevices; k++) {
    //     cudaDeviceProp prop;
    //     cudaGetDeviceProperties(&prop, k);
    //     std::cout << prop.name << std::endl;
    // }

    int n = 1<<20;
    int blockSize = 512;
    int numBlocks = (n + blockSize - 1) / blockSize;

    float *a, *b, *c;

    hipMallocManaged(&a, n * sizeof(float));
    hipMallocManaged(&b, n * sizeof(float));
    hipMallocManaged(&c, n * sizeof(float));

    populateVec<<<numBlocks, blockSize>>>(n, a, b);
    vecAdd<<<numBlocks, blockSize>>>(n, a, b, c);

    hipDeviceSynchronize();

    // for (int i = 0; i < 256; i++) {
    //     std::cout << c[i] << std::endl;
    // }

    return 0;

    // vecAdd<<<1, 256>>>()
}